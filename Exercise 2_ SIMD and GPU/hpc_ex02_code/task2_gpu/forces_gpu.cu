
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t result = call; \
    if (result != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \n", \
                __FILE__, __LINE__, static_cast<unsigned int>(result), hipGetErrorString(result)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

typedef struct Particle_s
{
  double x, y, z, m;
  double fx, fy, fz;
} Particle_t;

const double G = 6.67408e-11;

__global__ void computeGravitationalForcesKernel(Particle_t *particles, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        particles[i].fx = 0;
        particles[i].fy = 0;
        particles[i].fz = 0;

        for (int j = 0; j < n; j++)
        {
            if (i != j)
            {
                double tmp = pow(particles[i].x - particles[j].x, 2.0) +
                              pow(particles[i].y - particles[j].y, 2.0) +
                              pow(particles[i].z - particles[j].z, 2.0);

                double magnitude = G * particles[i].m * particles[j].m / pow(tmp, 1.5);

                particles[i].fx += (particles[i].x - particles[j].x) * magnitude;
                particles[i].fy += (particles[i].y - particles[j].y) * magnitude;
                particles[i].fz += (particles[i].z - particles[j].z) * magnitude;
            }
        }
    }
}

void computeGravitationalForces(Particle_t *particles, int n)
{
    Particle_t *d_particles;
    size_t size = n * sizeof(Particle_t);

    CUDA_CHECK(hipMalloc((void **)&d_particles, size));
    CUDA_CHECK(hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice));

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    computeGravitationalForcesKernel<<<gridSize, blockSize>>>(d_particles, n);

    CUDA_CHECK(hipMemcpy(particles, d_particles, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_particles));
}

// Rest of the code remains unchanged