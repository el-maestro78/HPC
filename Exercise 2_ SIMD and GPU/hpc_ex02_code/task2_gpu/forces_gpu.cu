#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define PRINTSTATS
/*#define CUDA_CHECK(call) \
do { \
    cudaError_t result = call; \
    if (result != cudaSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \n", \
                __FILE__, __LINE__, static_cast<unsigned int>(result), cudaGetErrorString(result)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)
*/
double get_wtime(void) {
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec*1.0e-6;
}

typedef struct Particle_s
{
  double x, y, z, m;
  double fx, fy, fz;
} Particle_t;

const double G = 6.67408e-11;

void initParticles(Particle_t *particles, int n)
{
  srand48(10);
  for (int i=0; i<n; i++) {
    particles[i].x = 10* drand48();
    particles[i].y = 10* drand48();
    particles[i].z = 10* drand48();
    particles[i].m = 1e7 / sqrt((double)n) *drand48();
  }
}

__global__ void computeGravitationalForcesKernel(Particle_t *particles, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        particles[i].fx = 0;
        particles[i].fy = 0;
        particles[i].fz = 0;

        for (int j = 0; j < n; j++)
        {
            if (i != j)
            {
                double tmp = pow(particles[i].x - particles[j].x, 2.0) +
                              pow(particles[i].y - particles[j].y, 2.0) +
                              pow(particles[i].z - particles[j].z, 2.0);

                double magnitude = G * particles[i].m * particles[j].m / pow(tmp, 1.5);

                particles[i].fx += (particles[i].x - particles[j].x) * magnitude;
                particles[i].fy += (particles[i].y - particles[j].y) * magnitude;
                particles[i].fz += (particles[i].z - particles[j].z) * magnitude;
            }
        }
    }
}

void computeGravitationalForces(Particle_t *particles, int n)
{
    Particle_t *d_particles;
    size_t size = n * sizeof(Particle_t);

//    CUDA_CHECK(cudaMalloc((void **)&d_particles, size));
//    CUDA_CHECK(cudaMemcpy(d_particles, particles, size, cudaMemcpyHostToDevice));

    hipMalloc((void **)&d_particles, size);
    hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice);
//    int blockSize = 256;
    dim3 blockSize(256);
//    int gridSize = (n + blockSize - 1) / blockSize;
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    computeGravitationalForcesKernel<<<gridSize, blockSize>>>(d_particles, n);

//    CUDA_CHECK(cudaMemcpy(particles, d_particles, size, cudaMemcpyDeviceToHost));
//    CUDA_CHECK(cudaFree(d_particles));
    hipMemcpy(particles, d_particles, size, hipMemcpyDeviceToHost);
    hipFree(d_particles);
}

void printStatistics(Particle_t *particles, int n)
{
  double sfx = 0, sfy = 0, sfz = 0;
  double maxfx = particles[0].fx;
  double minfx = particles[0].fx;
  double maxfy = particles[0].fy;
  double minfy = particles[0].fy;
  double maxfz = particles[0].fz;
  double minfz = particles[0].fz;
  for (int i=0; i<n; i++) {
    if (minfx < particles[i].fx) minfx = particles[i].fx;
    if (maxfx > particles[i].fx) maxfx = particles[i].fx;
    if (minfy < particles[i].fy) minfy = particles[i].fy;
    if (maxfy > particles[i].fy) maxfy = particles[i].fy;
    if (minfz < particles[i].fz) minfz = particles[i].fz;
    if (maxfz > particles[i].fz) maxfz = particles[i].fz;
    sfx += particles[i].fx;
    sfy += particles[i].fy;
    sfz += particles[i].fz;
  }

  printf("%d particles: sfx=%e sfy=%e sfz=%e\n", n, sfx, sfy, sfz);
  printf("%d particles: minfx=%f maxfx=%f\n", n, minfx, maxfx);
  printf("%d particles: minfy=%f maxfy=%f\n", n, minfy, maxfy);
  printf("%d particles: minfz=%f maxfz=%f\n", n, minfz, maxfz);
}


int main(int argc, char *argv[])
{
  int n;

  if (argc == 2)
    n = (1 << atoi(argv[1]));
  else
    n = (1 << 14);

  Particle_t *particles = (Particle_t *)malloc(n*sizeof(Particle_t));

  initParticles(particles, n);

  double t0 = get_wtime();
  computeGravitationalForces(particles, n);
  double t1 = get_wtime();

#if defined(PRINTSTATS)
  printStatistics(particles, n);
#endif

  printf("Elapsed time=%lf seconds\n", t1-t0);

  return 0;
}